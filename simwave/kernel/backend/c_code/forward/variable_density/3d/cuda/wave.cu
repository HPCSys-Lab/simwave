#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>
#include <omp.h>

#define THREADS_NUM_Y TY
#define THREADS_NUM_X TX
#define THREADS_NUM_Z TZ

#define checkErrorCuda(ans)                  \
    {                                        \
        gpuCheck((ans), __FILE__, __LINE__); \
    }

inline void gpuCheck(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPU Check: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
};

// use single (float) or double precision
// according to the value passed in the compilation cmd
#if defined(FLOAT)
typedef float f_type;
#elif defined(DOUBLE)
typedef double f_type;
#endif

__global__ void kernel_ExchangeValue(size_t nx, size_t nz, size_t ny,
                                     size_t current_t, size_t next_t,
                                     f_type *u)
{

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if ((i >= nz) || (j >= nx) || (k >= ny))
        return;

    size_t domain_size = nz * nx * ny;

    // index of the current point in the grid
    size_t domain_offset = (i * nx + j) * ny + k;

    size_t current_snapshot = current_t * domain_size + domain_offset;
    size_t next_snapshot = next_t * domain_size + domain_offset;

    f_type aux = u[current_snapshot];
    u[current_snapshot] = u[next_snapshot];
    u[next_snapshot] = aux;
}

__global__ void kernel_BoundaryCond_LeftRight(size_t nx, size_t nz, size_t ny,
                                              size_t z_before, size_t z_after,
                                              size_t y_before, size_t y_after,
                                              size_t x_before, size_t x_after,
                                              size_t next_t, size_t stencil_radius,
                                              f_type *u)
{

    // nz --> vertical
    // nx --> horizontal
    // ny --> diagonal

    int i = blockIdx.y * blockDim.y + threadIdx.y + stencil_radius;
    int j = blockIdx.x * blockDim.x + threadIdx.x + stencil_radius;

    if ((i >= (nz - stencil_radius)) || (j >= (nx - stencil_radius)))
        return;

    size_t domain_size = nz * nx * ny;

    // null dirichlet on the left
    if (y_before == 1)
    {
        size_t domain_offset = (i * nx + j) * ny + stencil_radius;
        size_t next_snapshot = next_t * domain_size + domain_offset;
        u[next_snapshot] = 0.0;
    }

    // null neumann on the left
    if (y_before == 2)
    {
        for (int ir = 1; ir <= stencil_radius; ir++)
        {
            size_t domain_offset = (i * nx + j) * ny + stencil_radius;
            size_t next_snapshot = next_t * domain_size + domain_offset;
            u[next_snapshot - ir] = u[next_snapshot + ir];
        }
    }

    // null dirichlet on the right
    if (y_after == 1)
    {
        size_t domain_offset = (i * nx + j) * ny + (ny - stencil_radius - 1);
        size_t next_snapshot = next_t * domain_size + domain_offset;
        u[next_snapshot] = 0.0;
    }

    // null neumann on the right
    if (y_after == 2)
    {
        for (int ir = 1; ir <= stencil_radius; ir++)
        {
            size_t domain_offset = (i * nx + j) * ny + (ny - stencil_radius - 1);
            size_t next_snapshot = next_t * domain_size + domain_offset;
            u[next_snapshot + ir] = u[next_snapshot - ir];
        }
    }
}

__global__ void kernel_BoundaryCond_FrontBack(size_t nx, size_t nz, size_t ny,
                                              size_t z_before, size_t z_after,
                                              size_t x_before, size_t x_after,
                                              size_t next_t, size_t stencil_radius,
                                              f_type *u)
{
    // nz --> vertical
    // nx --> horizontal
    // ny --> diagonal

    int i = blockIdx.y * blockDim.y + threadIdx.y + stencil_radius;
    int k = blockIdx.z * blockDim.z + threadIdx.z + stencil_radius;

    if ((i >= (nz - stencil_radius)) || (k >= (ny - stencil_radius)))
        return;

    size_t domain_size = nz * nx * ny;

    // null dirichlet on the front
    if (x_before == 1)
    {
        size_t domain_offset = (i * nx + stencil_radius) * ny + k;
        size_t next_snapshot = next_t * domain_size + domain_offset;
        u[next_snapshot] = 0.0;
    }

    // null neumann on the front
    if (x_before == 2)
    {
        for (int ir = 1; ir <= stencil_radius; ir++)
        {
            size_t domain_offset = (i * nx + stencil_radius) * ny + k;
            size_t next_snapshot = next_t * domain_size + domain_offset;
            u[next_snapshot - (ir * ny)] = u[next_snapshot + (ir * ny)];
        }
    }

    // null dirichlet on the back
    if (x_after == 1)
    {
        size_t domain_offset = (i * nx + (nx - stencil_radius - 1)) * ny + k;
        size_t next_snapshot = next_t * domain_size + domain_offset;
        u[next_snapshot] = 0.0;
    }

    // null neumann on the back
    if (x_after == 2)
    {
        for (int ir = 1; ir <= stencil_radius; ir++)
        {
            size_t domain_offset = (i * nx + (nx - stencil_radius - 1)) * ny + k;
            size_t next_snapshot = next_t * domain_size + domain_offset;
            u[next_snapshot + (ir * ny)] = u[next_snapshot - (ir * ny)];
        }
    }
}

__global__ void kernel_BoundaryCond_TopBottom(size_t nx, size_t nz, size_t ny,
                                              size_t z_before, size_t z_after,
                                              size_t x_before, size_t x_after,
                                              size_t next_t, size_t stencil_radius,
                                              f_type *u)
{

    int j = blockIdx.x * blockDim.x + threadIdx.x + stencil_radius;
    int k = blockIdx.z * blockDim.z + threadIdx.z + stencil_radius;

    if ((j >= (nx - stencil_radius)) || (k >= (ny - stencil_radius)))
        return;

    size_t domain_size = nz * nx * ny;

    // null dirichlet on the top
    if (z_before == 1)
    {
        size_t domain_offset = (stencil_radius * nx + j) * ny + k;
        size_t next_snapshot = next_t * domain_size + domain_offset;
        u[next_snapshot] = 0.0;
    }

    // null neumann on the top
    if (z_before == 2)
    {
        for (int ir = 1; ir <= stencil_radius; ir++)
        {
            size_t domain_offset = (stencil_radius * nx + j) * ny + k;
            size_t next_snapshot = next_t * domain_size + domain_offset;
            u[next_snapshot - (ir * nx * ny)] = u[next_snapshot + (ir * nx * ny)];
        }
    }

    // null dirichlet on the bottom
    if (z_after == 1)
    {
        size_t domain_offset = ((nz - stencil_radius - 1) * nx + j) * ny + k;
        size_t next_snapshot = next_t * domain_size + domain_offset;
        u[next_snapshot] = 0.0;
    }

    // null neumann on the bottom
    if (z_after == 2)
    {
        for (int ir = 1; ir <= stencil_radius; ir++)
        {
            size_t domain_offset = ((nz - stencil_radius - 1) * nx + j) * ny + k;
            size_t next_snapshot = next_t * domain_size + domain_offset;
            u[next_snapshot + (ir * nx * ny)] = u[next_snapshot - (ir * nx * ny)];
        }
    }
}

__global__ void kernel_ComputeReceive(size_t num_receivers, size_t *rec_points_interval,
                                      size_t *rec_points_values_offset, f_type *rec_points_values,
                                      size_t current_t, size_t nx, size_t nz, size_t ny,
                                      size_t n, f_type *receivers, f_type *u)
{

    int rec = blockIdx.x * blockDim.x + threadIdx.x;

    if (rec >= num_receivers)
        return;

    f_type sum = 0.0;
    size_t domain_size = nz * nx * ny;

    // each receiver has 6 (z_b, z_e, x_b, x_e, y_b, y_e) point intervals
    size_t offset_rec = rec * 6;

    // interval of grid points of the receiver in the Z axis
    size_t rec_z_begin = rec_points_interval[offset_rec + 0];
    size_t rec_z_end = rec_points_interval[offset_rec + 1];

    // interval of grid points of the receiver in the X axis
    size_t rec_x_begin = rec_points_interval[offset_rec + 2];
    size_t rec_x_end = rec_points_interval[offset_rec + 3];

    // interval of grid points of the receiver in the Y axis
    size_t rec_y_begin = rec_points_interval[offset_rec + 4];
    size_t rec_y_end = rec_points_interval[offset_rec + 5];

    // number of grid points of the receiver in each axis
    size_t rec_z_num_points = rec_z_end - rec_z_begin + 1;
    size_t rec_x_num_points = rec_x_end - rec_x_begin + 1;
    // size_t rec_y_num_points = rec_y_end - rec_y_begin + 1;

    // pointer to rec value offset
    size_t offset_rec_kws_index_z = rec_points_values_offset[rec];

    // index of the Kaiser windowed sinc value of the receiver point
    size_t kws_index_z = offset_rec_kws_index_z;

    // for each receiver point in the Z axis
    for (size_t i = rec_z_begin; i <= rec_z_end; i++)
    {
        size_t kws_index_x = offset_rec_kws_index_z + rec_z_num_points;

        // for each receiver point in the X axis
        for (size_t j = rec_x_begin; j <= rec_x_end; j++)
        {

            size_t kws_index_y = offset_rec_kws_index_z + rec_z_num_points + rec_x_num_points;

            // for each source point in the Y axis
            for (size_t k = rec_y_begin; k <= rec_y_end; k++)
            {

                f_type kws = rec_points_values[kws_index_z] * rec_points_values[kws_index_x] * rec_points_values[kws_index_y];

                // current receiver point in the grid
                size_t domain_offset = (i * nx + j) * ny + k;
                size_t current_snapshot = current_t * domain_size + domain_offset;
                sum += u[current_snapshot] * kws;

                kws_index_y++;
            }
            kws_index_x++;
        }
        kws_index_z++;
    }

    size_t current_rec_n = (n - 1) * num_receivers + rec;
    receivers[current_rec_n] = sum;
}

__global__ void kernel_AddSourceTerm(size_t n, size_t *src_points_interval, size_t src_points_interval_size,
                                     f_type *wavelet, size_t wavelet_size, size_t wavelet_count,
                                     size_t num_sources, f_type *src_points_values,
                                     size_t *src_points_values_offset,
                                     f_type *u, f_type *velocity, f_type dtSquared,
                                     size_t nx, size_t nz, size_t ny, size_t next_t)
{

    int src = blockIdx.x * blockDim.x + threadIdx.x;

    if (src >= num_sources)
        return;

    size_t domain_size = nz * nx * ny;
    size_t wavelet_offset = n - 1;

    if (wavelet_count > 1)
    {
        wavelet_offset = (n - 1) * num_sources + src;
    }

    if (wavelet[wavelet_offset] != 0.0)
    {

        // each source has 6 (z_b, z_e, x_b, x_e, y_b, y_e) point intervals
        size_t offset_src = src * 6;

        // interval of grid points of the source in the Z axis
        size_t src_z_begin = src_points_interval[offset_src + 0];
        size_t src_z_end = src_points_interval[offset_src + 1];

        // interval of grid points of the source in the X axis
        size_t src_x_begin = src_points_interval[offset_src + 2];
        size_t src_x_end = src_points_interval[offset_src + 3];

        // interval of grid points of the source in the Y axis
        size_t src_y_begin = src_points_interval[offset_src + 4];
        size_t src_y_end = src_points_interval[offset_src + 5];

        // number of grid points of the source in each axis
        size_t src_z_num_points = src_z_end - src_z_begin + 1;
        size_t src_x_num_points = src_x_end - src_x_begin + 1;
        // size_t src_y_num_points = src_y_end - src_y_begin + 1;

        // pointer to src value offset
        size_t offset_src_kws_index_z = src_points_values_offset[src];

        // index of the Kaiser windowed sinc value of the source point
        size_t kws_index_z = offset_src_kws_index_z;

        // for each source point in the Z axis
        for (size_t i = src_z_begin; i <= src_z_end; i++)
        {
            size_t kws_index_x = offset_src_kws_index_z + src_z_num_points;

            // for each source point in the X axis
            for (size_t j = src_x_begin; j <= src_x_end; j++)
            {

                size_t kws_index_y = offset_src_kws_index_z + src_z_num_points + src_x_num_points;

                // for each source point in the Y axis
                for (size_t k = src_y_begin; k <= src_y_end; k++)
                {

                    f_type kws = src_points_values[kws_index_z] * src_points_values[kws_index_x] * src_points_values[kws_index_y];

                    // current source point in the grid
                    size_t domain_offset = (i * nx + j) * ny + k;
                    size_t next_snapshot = next_t * domain_size + domain_offset;

                    f_type value = dtSquared * velocity[domain_offset] * velocity[domain_offset] * kws * wavelet[wavelet_offset];

                    // u[next_snapshot] += value;
                    atomicAdd(&u[next_snapshot], value);

                    kws_index_y++;
                }
                kws_index_x++;
            }
            kws_index_z++;
        }
    }
}

__global__ void kernel_UpdateWavefield(size_t prev_t, size_t current_t, size_t next_t,
                                       size_t stencil_radius, size_t nz, size_t nx, size_t ny, f_type dt,
                                       f_type dzSquared, f_type dxSquared, f_type dySquared, f_type dtSquared,
                                       f_type *u, f_type *velocity, f_type *density, f_type *coeff_order1, f_type *coeff_order2, f_type *damp)
{
    // nz --> vertical
    // nx --> horizontal
    // ny --> diagonal

    int i = blockIdx.y * blockDim.y + threadIdx.y + stencil_radius;
    int j = blockIdx.x * blockDim.x + threadIdx.x + stencil_radius;
    int k = blockIdx.z * blockDim.z + threadIdx.z + stencil_radius;

    if ((i >= (nz - stencil_radius)) || (j >= (nx - stencil_radius)) || (k >= (ny - stencil_radius)))
        return;

    size_t domain_size = nz * nx * ny;

    // index of the current point in the grid
    size_t domain_offset = (i * nx + j) * ny + k;

    size_t prev_snapshot = prev_t * domain_size + domain_offset;
    size_t current_snapshot = current_t * domain_size + domain_offset;
    size_t next_snapshot = next_t * domain_size + domain_offset;

    // stencil code to update grid
    f_type value = 0.0;

    // second derivative for pressure
    f_type sd_pressure_y = coeff_order2[0] * u[current_snapshot];
    f_type sd_pressure_x = coeff_order2[0] * u[current_snapshot];
    f_type sd_pressure_z = coeff_order2[0] * u[current_snapshot];

    // first derivative for pressure
    f_type fd_pressure_y = 0.0;
    f_type fd_pressure_x = 0.0;
    f_type fd_pressure_z = 0.0;

    // first derivative for density
    f_type fd_density_y = 0.0;
    f_type fd_density_x = 0.0;
    f_type fd_density_z = 0.0;

    // radius of the stencil
    for (int ir = 1; ir <= stencil_radius; ir++)
    {
        // neighbors in the Y direction
        sd_pressure_y += coeff_order2[ir] * (u[current_snapshot + ir] + u[current_snapshot - ir]);
        fd_pressure_y += coeff_order1[ir] * (u[current_snapshot + ir] - u[current_snapshot - ir]);
        fd_density_y += coeff_order1[ir] * (density[domain_offset + ir] - density[domain_offset - ir]);

        // neighbors in the X direction
        sd_pressure_x += coeff_order2[ir] * (u[current_snapshot + (ir * ny)] + u[current_snapshot - (ir * ny)]);
        fd_pressure_x += coeff_order1[ir] * (u[current_snapshot + (ir * nx)] - u[current_snapshot - (ir * nx)]);
        fd_density_x += coeff_order1[ir] * (density[domain_offset + (ir * nx)] - density[domain_offset - (ir * nx)]);

        // neighbors in the Z direction
        sd_pressure_z += coeff_order2[ir] * (u[current_snapshot + (ir * nx * ny)] + u[current_snapshot - (ir * nx * ny)]);
        fd_pressure_z += coeff_order1[ir] * (u[current_snapshot + (ir * nx * ny)] - u[current_snapshot - (ir * nx * ny)]);
        fd_density_z += coeff_order1[ir] * (density[domain_offset + (ir * nx * ny)] - density[domain_offset - (ir * nx * ny)]);
    }

    value += sd_pressure_y / dySquared + sd_pressure_x / dxSquared + sd_pressure_z / dzSquared;

    f_type term_y = (fd_pressure_y * fd_density_y) / (2 * dySquared);
    f_type term_x = (fd_pressure_x * fd_density_x) / (2 * dxSquared);
    f_type term_z = (fd_pressure_z * fd_density_z) / (2 * dzSquared);

    value -= (term_y + term_x + term_z) / density[domain_offset];

    // denominator with damp coefficient
    f_type denominator = (1.0 + damp[domain_offset] * dt);

    value *= (dtSquared * velocity[domain_offset] * velocity[domain_offset]) / denominator;

    u[next_snapshot] = 2.0 / denominator * u[current_snapshot] - ((1.0 - damp[domain_offset] * dt) / denominator) * u[prev_snapshot] + value;
}
// forward_2D_constant_density
extern "C" double forward(f_type *u, f_type *velocity, f_type *density, f_type *damp,
                          f_type *wavelet, size_t wavelet_size, size_t wavelet_count,
                          f_type *coeff_order2, f_type *coeff_order1, size_t *boundary_conditions,
                          size_t *src_points_interval, size_t src_points_interval_size,
                          f_type *src_points_values, size_t src_points_values_size,
                          size_t *src_points_values_offset,
                          size_t *rec_points_interval, size_t rec_points_interval_size,
                          f_type *rec_points_values, size_t rec_points_values_size,
                          size_t *rec_points_values_offset,
                          f_type *receivers, size_t num_sources, size_t num_receivers,
                          size_t nz, size_t nx, size_t ny, f_type dz, f_type dx, f_type dy,
                          size_t saving_stride, f_type dt,
                          size_t begin_timestep, size_t end_timestep,
                          size_t space_order, size_t num_snapshots)
{
    // nz --> vertical
    // nx --> horizontal
    // ny --> diagonal
    size_t stencil_radius = space_order / 2;

    size_t domain_size = nz * nx * ny;

    f_type dzSquared = dz * dz;
    f_type dxSquared = dx * dx;
    f_type dySquared = dy * dy;
    f_type dtSquared = dt * dt;

    // timestep pointers
    size_t prev_t = 0;
    size_t current_t = 1;
    size_t next_t = 2;

    // variable to measure execution time
    struct timeval time_start;
    struct timeval time_end;

    // get the start time
    gettimeofday(&time_start, NULL);

    // Device variables
    f_type *d_u;
    f_type *d_velocity;
    f_type *d_density;
    f_type *d_damp;
    f_type *d_coeff_order1;
    f_type *d_coeff_order2;
    size_t *d_src_points_interval;
    f_type *d_src_points_values;
    size_t *d_src_points_values_offset;
    size_t *d_rec_points_interval;
    f_type *d_rec_points_values;
    size_t *d_rec_points_values_offset;
    f_type *d_wavelet;
    f_type *d_receivers;

    // 1- Device memory allocation
    // 2- Data transfer from CPU to GPU memory

    long nbytes;
    size_t u_size = num_snapshots * domain_size;
    size_t shot_record_size = wavelet_size * num_receivers;

    nbytes = sizeof(f_type) * u_size;
    checkErrorCuda(hipMalloc(&d_u, nbytes));
    checkErrorCuda(hipMemcpy(d_u, u, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(f_type) * domain_size;
    checkErrorCuda(hipMalloc(&d_velocity, nbytes));
    checkErrorCuda(hipMemcpy(d_velocity, velocity, nbytes, hipMemcpyHostToDevice));

    checkErrorCuda(hipMalloc(&d_density, nbytes));
    checkErrorCuda(hipMemcpy(d_density, density, nbytes, hipMemcpyHostToDevice));

    checkErrorCuda(hipMalloc(&d_damp, nbytes));
    checkErrorCuda(hipMemcpy(d_damp, damp, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(f_type) * (stencil_radius + 1);
    checkErrorCuda(hipMalloc(&d_coeff_order2, nbytes));
    checkErrorCuda(hipMemcpy(d_coeff_order2, coeff_order2, nbytes, hipMemcpyHostToDevice));

    checkErrorCuda(hipMalloc(&d_coeff_order1, nbytes));
    checkErrorCuda(hipMemcpy(d_coeff_order1, coeff_order1, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(size_t) * src_points_interval_size;
    checkErrorCuda(hipMalloc(&d_src_points_interval, nbytes));
    checkErrorCuda(hipMemcpy(d_src_points_interval, src_points_interval, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(f_type) * src_points_values_size;
    checkErrorCuda(hipMalloc(&d_src_points_values, nbytes));
    checkErrorCuda(hipMemcpy(d_src_points_values, src_points_values, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(size_t) * num_sources;
    checkErrorCuda(hipMalloc(&d_src_points_values_offset, nbytes));
    checkErrorCuda(hipMemcpy(d_src_points_values_offset, src_points_values_offset, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(size_t) * rec_points_interval_size;
    checkErrorCuda(hipMalloc(&d_rec_points_interval, nbytes));
    checkErrorCuda(hipMemcpy(d_rec_points_interval, rec_points_interval, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(f_type) * rec_points_values_size;
    checkErrorCuda(hipMalloc(&d_rec_points_values, nbytes));
    checkErrorCuda(hipMemcpy(d_rec_points_values, rec_points_values, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(size_t) * num_receivers;
    checkErrorCuda(hipMalloc(&d_rec_points_values_offset, nbytes));
    checkErrorCuda(hipMemcpy(d_rec_points_values_offset, rec_points_values_offset, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(f_type) * wavelet_size * wavelet_count;
    checkErrorCuda(hipMalloc(&d_wavelet, nbytes));
    checkErrorCuda(hipMemcpy(d_wavelet, wavelet, nbytes, hipMemcpyHostToDevice));

    nbytes = sizeof(f_type) * shot_record_size;
    checkErrorCuda(hipMalloc(&d_receivers, nbytes));
    checkErrorCuda(hipMemcpy(d_receivers, receivers, nbytes, hipMemcpyHostToDevice));

    dim3 block = dim3(THREADS_NUM_X, THREADS_NUM_Y, THREADS_NUM_Z);
    dim3 grid;

    // wavefield modeling
    for (size_t n = begin_timestep; n <= end_timestep; n++)
    {

        // no saving case
        if (saving_stride == 0)
        {
            prev_t = (n - 1) % 3;
            current_t = n % 3;
            next_t = (n + 1) % 3;
        }
        else
        {
            // all timesteps saving case
            if (saving_stride == 1)
            {
                prev_t = n - 1;
                current_t = n;
                next_t = n + 1;
            }
        }

        /*
            Section 1: update the wavefield according to the acoustic wave equation
        */
        grid = dim3(ceilf(nx / (float)block.x), ceilf(nz / (float)block.y), ceilf(ny / (float)block.z));

        kernel_UpdateWavefield<<<grid, block>>>(prev_t, current_t, next_t,
                                                stencil_radius, nz, nx, ny, dt,
                                                dzSquared, dxSquared, dySquared, dtSquared,
                                                d_u, d_velocity, d_density, d_coeff_order1, d_coeff_order2, d_damp);

#if defined(DEBUG)
            checkErrorCuda(hipDeviceSynchronize());
#endif

        /*
            Section 2: add the source term
            NUM_SOURCE --> linear
        */
        grid = dim3(ceilf(num_sources / (float)block.x));

        kernel_AddSourceTerm<<<grid, block.x>>>(n, d_src_points_interval, src_points_interval_size,
                                                d_wavelet, wavelet_size, wavelet_count,
                                                num_sources, d_src_points_values,
                                                d_src_points_values_offset,
                                                d_u, d_velocity, dtSquared,
                                                nx, nz, ny, next_t);

#if defined(DEBUG)
        checkErrorCuda(hipDeviceSynchronize());
#endif

        /*
            Section 3: add boundary conditions (z_before, z_after, x_before, x_after)
            0 - no boundary condition
            1 - null dirichlet
            2 - null neumann
        */
        size_t z_before = boundary_conditions[0];
        size_t z_after = boundary_conditions[1];
        size_t x_before = boundary_conditions[2];
        size_t x_after = boundary_conditions[3];
        size_t y_before = boundary_conditions[4];
        size_t y_after = boundary_conditions[5];

        grid = dim3(ceilf(nx / (float)block.x), ceilf(nz / (float)block.y));

        kernel_BoundaryCond_LeftRight<<<grid, dim3(block.x, block.y)>>>(nx, nz, ny,
                                                                        z_before, z_after,
                                                                        y_before, y_after,
                                                                        x_before, x_after,
                                                                        next_t, stencil_radius,
                                                                        d_u);

#if defined(DEBUG)
        checkErrorCuda(hipDeviceSynchronize());
#endif
        grid = dim3(1, ceilf(nz / (float)block.y), ceilf(ny / (float)block.z));

        kernel_BoundaryCond_FrontBack<<<grid, dim3(1, block.y, block.z)>>>(nx, nz, ny,
                                                                           z_before, z_after,
                                                                           x_before, x_after,
                                                                           next_t, stencil_radius,
                                                                           d_u);

#if defined(DEBUG)
        checkErrorCuda(hipDeviceSynchronize());
#endif

        grid = dim3(ceilf(nx / (float)block.x), 1, ceilf(ny / (float)block.z));

        kernel_BoundaryCond_TopBottom<<<grid, dim3(block.x, 1, block.z)>>>(nx, nz, ny,
                                                                           z_before, z_after,
                                                                           x_before, x_after,
                                                                           next_t, stencil_radius,
                                                                           d_u);
#if defined(DEBUG)
        checkErrorCuda(hipDeviceSynchronize());
#endif

        /*
            Section 4: compute the receivers
        */
        grid = dim3(ceilf(num_receivers / (float)block.x));

        kernel_ComputeReceive<<<grid, block.x>>>(num_receivers, d_rec_points_interval,
                                                 d_rec_points_values_offset, d_rec_points_values,
                                                 current_t, nx, nz, ny,
                                                 n, d_receivers, d_u);

#if defined(DEBUG)
        checkErrorCuda(hipDeviceSynchronize());
#endif

        // stride timesteps saving case
        if (saving_stride > 1)
        {
            // shift the pointer
            if (n % saving_stride == 1)
            {

                prev_t = current_t;
                current_t += 1;
                next_t += 1;

                // even stride adjust case
                if (saving_stride % 2 == 0 && n < end_timestep)
                {
                    size_t swap = current_t;
                    current_t = next_t;
                    next_t = swap;

                    kernel_ExchangeValue<<<grid, block>>>(nx, nz, ny,
                                                          current_t, next_t,
                                                          d_u);

#if defined(DEBUG)
                    checkErrorCuda(hipDeviceSynchronize());
#endif
                }
            }
            else
            {
                prev_t = current_t;
                current_t = next_t;
                next_t = prev_t;
            }
        }
    }

    // Data transfer from GPU to CPU memory
    nbytes = sizeof(f_type) * u_size;
    checkErrorCuda(hipMemcpy(u, d_u, nbytes, hipMemcpyDeviceToHost));

    nbytes = sizeof(f_type) * shot_record_size;
    checkErrorCuda(hipMemcpy(receivers, d_receivers, nbytes, hipMemcpyDeviceToHost));

    // Free device memory
    checkErrorCuda(hipFree(d_u));
    checkErrorCuda(hipFree(d_velocity));
    checkErrorCuda(hipFree(d_damp));
    checkErrorCuda(hipFree(d_coeff_order1));
    checkErrorCuda(hipFree(d_coeff_order2));
    checkErrorCuda(hipFree(d_src_points_interval));
    checkErrorCuda(hipFree(d_src_points_values));
    checkErrorCuda(hipFree(d_src_points_values_offset));
    checkErrorCuda(hipFree(d_rec_points_interval));
    checkErrorCuda(hipFree(d_rec_points_values));
    checkErrorCuda(hipFree(d_rec_points_values_offset));
    checkErrorCuda(hipFree(d_wavelet));
    checkErrorCuda(hipFree(d_receivers));

    // get the end time
    gettimeofday(&time_end, NULL);

    double exec_time = (double)(time_end.tv_sec - time_start.tv_sec) + (double)(time_end.tv_usec - time_start.tv_usec) / 1000000.0;

    return exec_time;
}
